#include "hip/hip_runtime.h"
//
// Ray Tracing Gems sample code for 
//   "A Planetarium Dome Master Camera"
// 
// This code is a simplified derivative of the 
// C- and CUDA-based implementations in 
// the Tachyon ray tracing engine and the 
// VMD molecular visualization software.
// 
// Questions should be directed to the author
// John E. Stone, developer of Tachyon and VMD
//

#include "boilerplate.cuh"

//
// Camera ray generation code for planetarium dome display
// Generates a fisheye style frame with ~180 degree FoV
//
template<int STEREO_ON, int DOF_ON>
static __device__ __inline__
void camera_dome_general() {
  // Stereoscopic rendering is provided by rendering in an over/under
  // format with the left eye image into the top half of a double-high
  // framebuffer, and the right eye into the lower half.  The subsequent
  // OpenGL drawing code can trivially unpack and draw the two images
  // with simple pointer offset arithmetic.
  uint viewport_sz_y, viewport_idx_y;
  float eyeshift;
  if (STEREO_ON) {
    // render into a double-high framebuffer when stereo is enabled
    viewport_sz_y = launch_dim.y >> 1;
    if (launch_index.y >= viewport_sz_y) {
      // left image
      viewport_idx_y = launch_index.y - viewport_sz_y;
      eyeshift = -0.5f * cam_stereo_eyesep;
    } else {
      // right image
      viewport_idx_y = launch_index.y;
      eyeshift =  0.5f * cam_stereo_eyesep;
    }
  } else {
    // render into a normal size framebuffer if stereo is not enabled
    viewport_sz_y = launch_dim.y;
    viewport_idx_y = launch_index.y;
    eyeshift = 0.0f;
  }

  float fov = 180.0f;                          // dome FoV in degrees

  // half FoV in radians, pixels beyond this distance are outside
  // of the field of view of the projection, and are set black
  float rmax = 0.5 * fov * (M_PIf / 180.0f);

  // The dome angle from center of the projection is proportional
  // to the image-space distance from the center of the viewport.
  // viewport_sz contains the viewport size, radperpix contains the
  // radians/pixel scaling factors in X/Y, and viewport_mid contains
  // the midpoint coordinate of the viewpoint used to compute the
  // distance from center.
  float2 viewport_sz = make_float2(launch_dim.x, viewport_sz_y);
  float2 radperpix = (M_PIf / 180.0f) * fov / viewport_sz;
  float2 viewport_mid = viewport_sz * 0.5f;

  unsigned int randseed = tea<4>(launch_dim.x*(launch_index.y)+launch_index.x, subframe_count());

  float3 col = make_float3(0.0f);
  float alpha = 0.0f;
  for (int s=0; s<aa_samples; s++) {
    // compute the jittered image plane sample coordinate
    float2 jxy;
    jitter_offset2f(randseed, jxy);
    float2 viewport_idx = make_float2(launch_index.x, viewport_idx_y) + jxy;

    // compute the ray angles in X/Y and total angular distance from center
    float2 rd = (viewport_idx - viewport_mid) * radperpix;
    float rangle = hypotf(rd.x, rd.y);

    // pixels outside the dome FoV are treated as black by not
    // contributing to the color accumulator
    if (rangle < rmax) {
      float3 ray_direction;
      float3 ray_origin = cam_pos;

      if (rangle == 0) {
        // handle center of dome where azimuth is undefined by
        // setting the ray direction to the zenith
        ray_direction = cam_W;
      } else {
        float rasin, racos;
        sincosf(rangle, &rasin, &racos);
        float rsin = rasin / rangle;
        float rcos = racos / rangle;
        ray_direction = cam_U*rsin*rd.x + cam_V*rsin*rd.y + cam_W*racos;
        up_direction = -cam_U*rcos*rd.x - cam_V*rcos*rd.y + cam_W*rasin;
        right_direction = cam_U*(rd.y/rangle) + cam_V*(-rd.x/rangle);

        if (STEREO_ON) {
          // assumes a flat dome, where cam_W also points in the 
          // audience "up" direction
          ray_origin += eyeshift * powf(cross(ray_direction, cam_W), 5.0);
        }

        if (DOF_ON) {
          dof_ray(ray_origin, ray_origin, ray_direction, ray_direction,
                  randseed, up_direction, right_direction);
        }
      }

      // trace the new ray...
      PerRayData_radiance prd;
      prd.importance = 1.f;
      prd.alpha = 1.f;
      prd.depth = 0;
      prd.transcnt = max_trans;
      optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(root_object, ray, prd);
      col += prd.result;
      alpha += prd.alpha;
    }
  }

  accumulate_color(col, alpha);
}


//
// Template instantiations to create optimized
// case-specific versions of the raygen program.
//
RT_PROGRAM void camera_dome_master() {
  camera_dome_general<0, 0>();
}

RT_PROGRAM void camera_dome_master_dof() {
  camera_dome_general<0, 1>();
}

RT_PROGRAM void camera_dome_master_stereo() {
  camera_dome_general<1, 0>();
}

RT_PROGRAM void camera_dome_master_stereo_dof() {
  camera_dome_general<1, 1>();
}



