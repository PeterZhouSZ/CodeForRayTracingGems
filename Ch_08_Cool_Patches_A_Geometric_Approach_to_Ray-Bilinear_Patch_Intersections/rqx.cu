#include "hip/hip_runtime.h"
RT_PROGRAM void intersectPatch(int prim_idx) {
	// ray is rtDeclareVariable(Ray,ray,rtCurrentRay,) in OptiX
	const PatchData& patch = patchdata[prim_idx]; // patchdata is optix::rtBuffer
	const float3* q = patch.coefficients();       // 4 corners + "normal" qn
	float3 q00 = q[0], q10 = q[1], q11 = q[2], q01 = q[3];
	float3 e10 = q10 - q00; // q01---------------q11
	float3 e11 = q11 - q10; // |                   |
	float3 e00 = q01 - q00; // | e00           e11 |  we precompute
	float3 qn  = q[4];      // |        e10        |  qn = cross(q10-q00,q01-q11)
	q00 -= ray.origin;      // q00---------------q10
	q10 -= ray.origin;
	float a = dot(cross(q00, ray.direction), e00); // the equation is /*\label{code:a}*/
	float c = dot(qn, ray.direction);              // a + b u + c u^2 /*\label{code:c}*/
	float b = dot(cross(q10, ray.direction), e11); // first compute a+b+c
	b -= a + c;                                    // and then b /*\label{code:b}*/
	float det = b*b - 4*a*c;
	if (det < 0) return;      // see the right part of Figure 8.5
	det = sqrt(det);          // we -use_fast_math in CUDA_NVRTC_OPTIONS
	float u1, u2;             // two roots (u parameter)
	float t = ray.tmax, u, v; // need solution for the smallest t > 0  
	if (c == 0) {                        // if c == 0, it is a trapezoid /*\label{code:t}*/
		u1  = -a/b; u2 = -1;              // and there is only one root
	} else {                             // (c != 0 in Stanford models)
		u1  = (-b - copysignf(det, b))/2; // numerically "stable" root           /*\label{code:u1}*/
		u2  = a/u1;                       // Viete's formula for u1*u2
		u1 /= c;
	}
	if (0 <= u1 && u1 <= 1) {                // is it inside the patch?
		float3 pa = lerp(q00, q10, u1);       // point on edge e10 (Figure 8.4)
		float3 pb = lerp(e00, e11, u1);       // it is, actually, pb - pa
		float3 n  = cross(ray.direction, pb);
		det = dot(n, n);
		n = cross(n, pa);
		float t1 = dot(n, pb);
		float v1 = dot(n, ray.direction);     // no need to check t1 < t		
		if (t1 > 0 && 0 <= v1 && v1 <= det) { // if t1 > ray.tmax, 					
			t = t1/det; u = u1; v = v1/det;    // it will be rejected				
		}                                     // in rtPotentialIntersection
	}
	if (0 <= u2 && u2 <= 1) {                // it is slightly different,
		float3 pa = lerp(q00, q10, u2);       // since u1 might be good /*\label{code:v2}*/
		float3 pb = lerp(e00, e11, u2);       // and we need 0 < t2 < t1
		float3 n  = cross(ray.direction, pb);
		det = dot(n, n);
		n = cross(n, pa);
		float t2 = dot(n, pb)/det;
		float v2 = dot(n, ray.direction);
		if (0 <= v2 && v2 <= det && t > t2 && t2 > 0) {
			t = t2; u = u2; v = v2/det;
		}
	}
	if (rtPotentialIntersection(t)) {
		// Fill the intersection structure irec.
		// Normal(s) for the closest hit will be normalized in a shader.
		float3 du = lerp(e10, q11 - q01, v);
		float3 dv = lerp(e00, e11, u);
		irec.geometric_normal = cross(du, dv);
		#if defined(SHADING_NORMALS)
		const float3* vn = patch.vertex_normals;
		irec.shading_normal = lerp(lerp(vn[0],vn[1],u), 
		                           lerp(vn[3],vn[2],u),v);
		#else
		irec.shading_normal = irec.geometric_normal;
		#endif
		irec.texcoord = make_float3(u, v, 0);
		irec.id = prim_idx;
		rtReportIntersection(0u);
	}
}
