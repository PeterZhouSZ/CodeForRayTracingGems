#include "hip/hip_runtime.h"
// Implementation of our method as described in Section 6.2.2.

constexpr float origin()      { return 1.0f / 32.0f; }
constexpr float float_scale() { return 1.0f / 65536.0f; }
constexpr float int_scale()   { return 256.0f; }

// Normal points outward for rays exiting the surface, else is flipped.
float3 offset_ray(const float3 p, const float3 n)
{
  int3 of_i(int_scale() * n.x, int_scale() * n.y, int_scale() * n.z);

  float3 p_i(
     int_as_float(float_as_int(p.x) + ((p.x < 0) ? -of_i.x : of_i.x)),
     int_as_float(float_as_int(p.y) + ((p.y < 0) ? -of_i.y : of_i.y)),
     int_as_float(float_as_int(p.z) + ((p.z < 0) ? -of_i.z : of_i.z)));

  return float3(fabsf(p.x) < origin() ? p.x+float_scale()*n.x : p_i.x,
                fabsf(p.y) < origin() ? p.y+float_scale()*n.y : p_i.y,
                fabsf(p.z) < origin() ? p.z+float_scale()*n.z : p_i.z);
}